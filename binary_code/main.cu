#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "pibr.h"

int main(int argc, char *argv[])
{
    if (argc != 5)
    {
        printf("Give arguments: image, height, width, max_order\n");
        return 1;
    }

    char filename[100];
    int height, width, max_order, *dim_order, *device_dim_order;
    strcpy(filename, argv[1]);
    height = atoi(argv[2]);
    width = atoi(argv[3]);
    max_order = atoi(argv[4]);

    float kernelElapsedTime = 0;
    float kernelElapsedTime1 = 0;
    float kernelElapsedTime11 = 0;
    float kernelElapsedTime_poly = 0;
    float kernelElapsedTime_cheby_block_mom = 0;
    float kernelElapsedTime_block_moment = 0;
    hipEvent_t start, stop, start11, stop11, start111, stop111, start_poly, stop_poly, start_block_moment, stop_block_moment, start_cheby_block_mom, stop_cheby_block_mom;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start11);
    hipEventCreate(&stop11);
    hipEventCreate(&start111);
    hipEventCreate(&stop111);
    hipEventCreate(&start_poly);
    hipEventCreate(&stop_poly);
    hipEventCreate(&start_cheby_block_mom);
    hipEventCreate(&stop_cheby_block_mom);

    hipEventCreate(&start_block_moment);
    hipEventCreate(&stop_block_moment);
    struct timeval tim;
    double start1, stop1, start2, stop2;
    int *host_interval_row, *device_interval_row, *host_check, *device_check, *device_blockno, *host_blockno, blockno = 0;
    unsigned char *host_image, *device_image, **blockimg;
    introw_t *device_ir;
    blocktype *host_block, *device_block, *host_a, *device_a;
    dim_order = (int *)malloc(2 * sizeof(int));
    dim_order[0] = MAX(height, width); // MAX
    dim_order[1] = max_order;

    size_t bytes = height * width * sizeof(unsigned char);

    float *host_poly, *device_poly, *reconstructed_image;
    float *device_ttx, *device_tb, *device_r;
    float *device_stx, *device_sty, *host_tb, *host_t2d;

    host_poly = (float *)malloc(dim_order[0] * dim_order[1] * sizeof(float));
    host_tb = (float *)calloc(dim_order[1] * dim_order[1], sizeof(float));
    host_t2d = (float *)malloc(dim_order[1] * dim_order[1] * sizeof(float));
    host_blockno = (int *)malloc(height * (width / 2) * sizeof(int));
    reconstructed_image = (float *)malloc(dim_order[0] * dim_order[0] * sizeof(float));
    host_a = (blocktype *)malloc(height * width * sizeof(blocktype));
    host_check = (int *)malloc(height * width * sizeof(int));
    host_block = (blocktype *)malloc(height * width * sizeof(blocktype));
    host_image = (unsigned char *)malloc(bytes);
    host_interval_row = (int *)malloc(height * sizeof(int));

    gpuErrchk(hipMalloc(&device_dim_order, 2 * sizeof(int)));
    gpuErrchk(hipMalloc(&device_a, height * width * sizeof(blocktype)));
    gpuErrchk(hipMalloc(&device_blockno, height * (width / 2) * sizeof(int)));
    gpuErrchk(hipMalloc((void **)&device_poly, dim_order[0] * dim_order[1] * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&device_ttx, (dim_order[1] * dim_order[0] + dim_order[0]) * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&device_r, dim_order[1] * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&device_tb, dim_order[1] * dim_order[1] * sizeof(float)));
    gpuErrchk(hipMalloc(&device_block, height * width * sizeof(blocktype)));
    gpuErrchk(hipMalloc(&device_check, height * width * sizeof(int)));
    gpuErrchk(hipMalloc(&device_ir, height * width * sizeof(introw_t)));
    gpuErrchk(hipMalloc(&device_image, bytes));
    gpuErrchk(hipMalloc(&device_interval_row, height * sizeof(int)));

    read(filename, host_image, height, width);

    gettimeofday(&tim, NULL);
    start1 = (tim.tv_sec * 1000.0 + (tim.tv_usec / 1000.0));
    gpuErrchk(hipMemcpy(device_image, host_image, bytes, hipMemcpyHostToDevice));
    gettimeofday(&tim, NULL);
    stop1 = (tim.tv_sec * 1000.0 + (tim.tv_usec / 1000.0));

    gpuErrchk(hipMemcpy(device_dim_order, dim_order, 2 * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_interval_row, host_interval_row, height * sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipEventRecord(start));
    pibr_extraction<<<GRID_SIZE, BLOCK_SIZE>>>(device_image, height, width, device_interval_row, device_ir);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    gpuErrchk(hipEventElapsedTime(&kernelElapsedTime, start, stop));

    hipFree(device_image);

    gpuErrchk(hipEventRecord(start11));
    pibr_block_creation<<<GRID_SIZE, BLOCK_SIZE>>>(device_ir, height, width, device_block, device_interval_row, device_check, device_blockno);
    hipEventRecord(stop11);
    hipEventSynchronize(stop11);
    gpuErrchk(hipEventElapsedTime(&kernelElapsedTime1, start11, stop11));

    gpuErrchk(hipEventRecord(start111));
    pibr_block_creation2<<<GRID_SIZE, BLOCK_SIZE>>>(device_ir, height, width, device_block, device_interval_row, device_check, device_blockno);
    hipEventRecord(stop111);
    hipEventSynchronize(stop111);
    gpuErrchk(hipEventElapsedTime(&kernelElapsedTime11, start111, stop111));

    gpuErrchk(hipEventRecord(start_poly));
    cheby_poly<<<GRID_SIZE, BLOCK_SIZE>>>(device_poly, device_dim_order);
    hipEventRecord(stop_poly);
    hipEventSynchronize(stop_poly);
    gpuErrchk(hipEventElapsedTime(&kernelElapsedTime_poly, start_poly, stop_poly));

    gpuErrchk(hipMemcpy(host_interval_row, device_interval_row, height * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(host_block, device_block, height * width * sizeof(blocktype), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(host_blockno, device_blockno, height * (width / 2) * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(host_check, device_check, height * width * sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(host_poly, device_poly, dim_order[0] * dim_order[1] * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(device_block);
    hipFree(device_check);
    hipFree(device_poly);
    hipFree(device_ir);
    hipFree(device_interval_row);

    write(dim_order[0], 0, host_block, host_interval_row, host_blockno);

    convert_block_indexing(host_block, host_check, height, width, host_a, host_interval_row, blockno, host_blockno);
    gpuErrchk(hipMemcpy(device_a, host_a, height * width * sizeof(int), hipMemcpyHostToDevice));

    blockimg = (unsigned char **)malloc(height * sizeof(unsigned char *));
    for (int i = 0; i < height; i++)
    {
        blockimg[i] = (unsigned char *)malloc(width * sizeof(int));
        if (!blockimg[i])
        {
            printf("not enough memory\n");
            exit(1);
        }
    }
    int total = 0;
    for (int i = 0; i < height; i++)
    {
        total += host_interval_row[i];
    }

    write_poly(host_poly, dim_order);

    int total_blocks = 0;
    for (int i = 0; i < height; i++)
    {
        for (int y = 0; y < host_interval_row[i]; y++)
        {
            if (host_check[(i * height + y)] == 1)
            {
                total_blocks++;
            }
        }
    }

    gpuErrchk(hipMalloc((void **)&device_stx, dim_order[1] * total * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&device_sty, dim_order[1] * total * sizeof(float)));

    hipEventRecord(start_cheby_block_mom);
    tt_calculation<<<GRID_SIZE, BLOCK_SIZE>>>(device_dim_order, device_a, device_ttx, device_stx, device_sty);
    hipEventRecord(stop_cheby_block_mom);
    hipEventSynchronize(stop_cheby_block_mom);
    gpuErrchk(hipEventElapsedTime(&kernelElapsedTime_cheby_block_mom, start_cheby_block_mom, stop_cheby_block_mom));

    hipEventRecord(start_block_moment);
    tb_calculation<<<total_blocks, BLOCK_SIZE>>>(device_dim_order, device_a, device_ttx, device_stx, device_sty, total_blocks, device_tb, device_r);
    hipEventRecord(stop_block_moment);
    hipEventSynchronize(stop_block_moment);
    gpuErrchk(hipEventElapsedTime(&kernelElapsedTime_block_moment, start_block_moment, stop_block_moment));

    gettimeofday(&tim, NULL);
    start2 = (tim.tv_sec * 1000.0 + (tim.tv_usec / 1000.0));
    gpuErrchk(hipMemcpy(host_tb, device_tb, dim_order[1] * dim_order[1] * sizeof(float), hipMemcpyDeviceToHost));
    gettimeofday(&tim, NULL);
    stop2 = (tim.tv_sec * 1000.0 + (tim.tv_usec / 1000.0));

    // cheby_rebuild(height, width, dim_order, reconstructed_image, host_poly, host_tb);
    // write1d(reconstructed_image, height, width);

    double nire = nire_calculation(host_image, reconstructed_image, height, width);
    // make block image from extracted blocks
    // make_block_image(width, height, blockimg, host_block, host_check, host_interval_row);
    // write2d(blockimg, height, width);

    printf("total blocks %d\n", total_blocks);
    printf("intervals = %d\n\n", total);
    printf("data transfers time = %f msec\n\n", (stop1 - start1) + (stop2 - start2));
    printf("interval extraction = %1.4f msec\n", kernelElapsedTime);
    printf("interval matching = %1.4f msec\n\n", kernelElapsedTime1);
    printf("block extraction = %1.4f msec\n\n", kernelElapsedTime11);
    printf("poly processing time = %f msec\n", kernelElapsedTime_poly);
    printf("modified poly processing time = %f msec\n", kernelElapsedTime_cheby_block_mom);
    printf("BLOCK MOMENT processing time = %f msec\n", kernelElapsedTime_block_moment + kernelElapsedTime_cheby_block_mom);
    printf("NIRE = %f \n", nire);

    hipFree(device_stx);
    hipFree(device_sty);
    hipFree(device_ttx);
    hipFree(device_dim_order);
    hipFree(device_a);
    hipFree(device_r);
    hipFree(device_tb);
    free(host_interval_row);
    free(host_check);
    free(host_poly);
    free(host_image);
    free(host_t2d);
    free(host_tb);
    free(dim_order);
    free(host_block);
    free(host_a);

    return 0;
}
